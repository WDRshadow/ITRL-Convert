#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

#include "formatting.h"

#define Y_R 16896 // 66 * 256
#define Y_G 33024 // 129 * 256
#define Y_B 6400  // 25 * 256

#define U_R -9728  // -38 * 256
#define U_G -18944 // -74 * 256
#define U_B 28672  // 112 * 256

#define V_R 28672  // 112 * 256
#define V_G -24064 // -94 * 256
#define V_B -4608  // -18 * 256

#define CLAMP(x) ((x) < 0 ? 0 : ((x) > 255 ? 255 : x))

#define BLOCK_SIZE 32, 16

__global__ void rgb2yuyv_kernel(const unsigned char *rgb, unsigned char *yuyv, unsigned int width, unsigned int height)
{
    int x = (blockIdx.x * blockDim.x + threadIdx.x) * 2;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x + 1 < width && y < height)
    {
        int index_rgb = (y * width + x) * 3;
        int index_yuyv = (y * width + x) * 2;

        unsigned char r0 = rgb[index_rgb];
        unsigned char g0 = rgb[index_rgb + 1];
        unsigned char b0 = rgb[index_rgb + 2];
        unsigned char r1 = rgb[index_rgb + 3];
        unsigned char g1 = rgb[index_rgb + 4];
        unsigned char b1 = rgb[index_rgb + 5];

        unsigned char y0 = CLAMP(((Y_R * r0 + Y_G * g0 + Y_B * b0) >> 16) + 16);
        unsigned char y1 = CLAMP(((Y_R * r1 + Y_G * g1 + Y_B * b1) >> 16) + 16);
        unsigned char u = CLAMP(((((U_R * r0 + U_G * g0 + U_B * b0) + (U_R * r1 + U_G * g1 + U_B * b1)) / 2) >> 16) + 128);
        unsigned char v = CLAMP(((((V_R * r0 + V_G * g0 + V_B * b0) + (V_R * r1 + V_G * g1 + V_B * b1)) / 2) >> 16) + 128);

        yuyv[index_yuyv] = y0;
        yuyv[index_yuyv + 1] = u;
        yuyv[index_yuyv + 2] = y1;
        yuyv[index_yuyv + 3] = v;
    }
}

__global__ void bgra2rgb_kernel(const unsigned char *bgra, unsigned char *rgb, unsigned int width, unsigned int height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height)
    {
        int index_bgra = (y * width + x) * 4;
        int index_rgb = (y * width + x) * 3;

        rgb[index_rgb] = bgra[index_bgra + 2];     // R (from BGRA position 2)
        rgb[index_rgb + 1] = bgra[index_bgra + 1]; // G (from BGRA position 1)
        rgb[index_rgb + 2] = bgra[index_bgra];     // B (from BGRA position 0)
    }
}

unsigned char *get_cuda_buffer(size_t size)
{
    unsigned char *h_pinned = nullptr;
    hipHostMalloc((void **)&h_pinned, size);
    return h_pinned;
}

void free_cuda_buffer(unsigned char *h_pinned)
{
    if (h_pinned)
    {
        hipHostFree(h_pinned);
    }
}

CudaImageConverter::CudaImageConverter(unsigned int width, unsigned int height, int stream_num, int mode)
    : width(width), height(height), stream_num(stream_num),
      block_height(height / stream_num),
      size_bgra_block(width * block_height * 4),
      size_yuyv_block(width * block_height * 2),
      size_rgb_block(width * block_height * 3),
      mode(mode)
{
    blockSize = new dim3(BLOCK_SIZE);
    streams = new hipStream_t[stream_num];
    for (int i = 0; i < stream_num; i++)
    {
        hipStreamCreate(&streams[i]);
    }
    if (mode == D_BGRA2RGB)
    {
        gridSize = new dim3((width + blockSize->x - 1) / blockSize->x, (block_height + blockSize->y - 1) / blockSize->y);
    }
    else if (mode == RGB2YUYV)
    {
        gridSize = new dim3((width / 2 + blockSize->x - 1) / blockSize->x, (block_height + blockSize->y - 1) / blockSize->y);
        hipMalloc((void **)&d_yuyv, width * height * 2);
    }
    hipMalloc((void **)&d_rgb, width * height * 3);
}

CudaImageConverter::~CudaImageConverter()
{
    for (int i = 0; i < stream_num; i++)
    {
        hipStreamDestroy(streams[i]);
    }
    free(streams);
    delete blockSize;
    delete gridSize;
    if (mode == RGB2YUYV)
        hipFree(d_yuyv);
    hipFree(d_rgb);
}

void CudaImageConverter::convert(const unsigned char *src, unsigned char *dst)
{
    for (int i = 0; i < stream_num; i++)
    {
        if (mode == D_BGRA2RGB)
        {
            bgra2rgb_kernel<<<*gridSize, *blockSize, 0, streams[i]>>>(
                src + i * size_bgra_block,
                d_rgb + i * size_rgb_block,
                width,
                block_height);

            hipMemcpyAsync(
                dst + i * size_rgb_block,
                d_rgb + i * size_rgb_block,
                size_rgb_block,
                hipMemcpyDeviceToHost,
                streams[i]);
        }
        else if (mode == RGB2YUYV)
        {
            hipMemcpyAsync(
                d_rgb + i * size_rgb_block,
                src + i * size_rgb_block,
                size_rgb_block,
                hipMemcpyHostToDevice,
                streams[i]);

            rgb2yuyv_kernel<<<*gridSize, *blockSize, 0, streams[i]>>>(
                d_rgb + i * size_rgb_block,
                d_yuyv + i * size_yuyv_block,
                width,
                block_height);

            hipMemcpyAsync(
                dst + i * size_yuyv_block,
                d_yuyv + i * size_yuyv_block,
                size_yuyv_block,
                hipMemcpyDeviceToHost,
                streams[i]);
        }
    }
    hipDeviceSynchronize();
}
