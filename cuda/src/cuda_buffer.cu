#include <hip/hip_runtime.h>

#include "cuda_buffer.h"

unsigned char *get_cuda_buffer(size_t size)
{
    unsigned char *h_pinned = nullptr;
    hipHostMalloc((void **)&h_pinned, size);
    return h_pinned;
}

void free_cuda_buffer(unsigned char *h_pinned)
{
    if (h_pinned)
    {
        hipHostFree(h_pinned);
    }
}