#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

#include "rgb2yuyv.h"

#define Y_R 16896 // 66 * 256
#define Y_G 33024 // 129 * 256
#define Y_B 6400  // 25 * 256

#define U_R -9728  // -38 * 256
#define U_G -18944 // -74 * 256
#define U_B 28672  // 112 * 256

#define V_R 28672  // 112 * 256
#define V_G -24064 // -94 * 256
#define V_B -4608  // -18 * 256

#define CLAMP(x) ((x) < 0 ? 0 : ((x) > 255 ? 255 : x))

bool is_cuda_initialized = false;
int stream_num_;
hipStream_t *streams = nullptr;
unsigned int width_;
unsigned int height_;
unsigned int block_height;
size_t size_rgb24_block;
size_t size_yuyv422_block;
unsigned char *d_rgb24 = nullptr;
unsigned char *d_yuyv422 = nullptr;

const dim3 blockSize(32, 16);
dim3 gridSize;

__global__ void rgb2yuyv_kernel(const unsigned char *rgb24, unsigned char *yuyv422, unsigned int width, unsigned int height)
{
    int x = (blockIdx.x * blockDim.x + threadIdx.x) * 2;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height)
    {
        int index_rgb = (y * width + x) * 3;
        int index_yuyv = (y * width + x) * 2;

        unsigned char r0 = rgb24[index_rgb];
        unsigned char g0 = rgb24[index_rgb + 1];
        unsigned char b0 = rgb24[index_rgb + 2];
        unsigned char r1 = rgb24[index_rgb + 3];
        unsigned char g1 = rgb24[index_rgb + 4];
        unsigned char b1 = rgb24[index_rgb + 5];

        unsigned char y0 = CLAMP(((Y_R * r0 + Y_G * g0 + Y_B * b0) >> 16) + 16);
        unsigned char y1 = CLAMP(((Y_R * r1 + Y_G * g1 + Y_B * b1) >> 16) + 16);
        unsigned char u = CLAMP(((((U_R * r0 + U_G * g0 + U_B * b0) + (U_R * r1 + U_G * g1 + U_B * b1)) / 2) >> 16) + 128);
        unsigned char v = CLAMP(((((V_R * r0 + V_G * g0 + V_B * b0) + (V_R * r1 + V_G * g1 + V_B * b1)) / 2) >> 16) + 128);

        yuyv422[index_yuyv] = y0;
        yuyv422[index_yuyv + 1] = u;
        yuyv422[index_yuyv + 2] = y1;
        yuyv422[index_yuyv + 3] = v;
    }
}

void init_rgb2yuyv_cuda(unsigned int width, unsigned int height, int stream_num)
{
    if (is_cuda_initialized)
    {
        cleanup_rgb2yuyv_cuda();
    }
    width_ = width;
    height_ = height;
    stream_num_ = stream_num;
    block_height = height / stream_num;
    size_rgb24_block = width * block_height * 3;
    size_yuyv422_block = width * block_height * 2;
    gridSize = dim3((width / 2 + blockSize.x - 1) / blockSize.x, (block_height + blockSize.y - 1) / blockSize.y);
    streams = (hipStream_t *)malloc(stream_num * sizeof(hipStream_t));
    for (int i = 0; i < stream_num; i++)
    {
        hipStreamCreate(&streams[i]);
    }
    hipMalloc((void **)&d_rgb24, width * height * 3);
    hipMalloc((void **)&d_yuyv422, width * height * 2);
    is_cuda_initialized = true;
}

void rgb2yuyv_cuda(const unsigned char *rgb24, unsigned char *yuyv422)
{
    if (!is_cuda_initialized)
    {
        return;
    }
    for (int i = 0; i < stream_num_; i++)
    {
        hipMemcpyAsync(
            d_rgb24 + i * size_rgb24_block,
            rgb24 + i * size_rgb24_block,
            size_rgb24_block,
            hipMemcpyHostToDevice,
            streams[i]);

        rgb2yuyv_kernel<<<gridSize, blockSize, 0, streams[i]>>>(
            d_rgb24 + i * size_rgb24_block,
            d_yuyv422 + i * size_yuyv422_block,
            width_,
            block_height);

        hipMemcpyAsync(
            yuyv422 + i * size_yuyv422_block,
            d_yuyv422 + i * size_yuyv422_block,
            size_yuyv422_block,
            hipMemcpyDeviceToHost,
            streams[i]);
    }
    hipDeviceSynchronize();
}

void cleanup_rgb2yuyv_cuda()
{
    if (is_cuda_initialized)
    {
        for (int i = 0; i < stream_num_; i++)
        {
            hipStreamDestroy(streams[i]);
        }
        free(streams);
        streams = nullptr;
        hipFree(d_rgb24);
        d_rgb24 = nullptr;
        hipFree(d_yuyv422);
        d_yuyv422 = nullptr;
        is_cuda_initialized = false;
    }
}

unsigned char *get_cuda_buffer(size_t size)
{
    unsigned char *h_pinned = nullptr;
    hipHostMalloc((void **)&h_pinned, size);
    return h_pinned;
}

void free_cuda_buffer(unsigned char *h_pinned)
{
    if (h_pinned)
    {
        hipHostFree(h_pinned);
    }
}
