#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

#include "rgb2yuyv.h"

#define Y_R 16896 // 66 * 256
#define Y_G 33024 // 129 * 256
#define Y_B 6400  // 25 * 256

#define U_R -9728  // -38 * 256
#define U_G -18944 // -74 * 256
#define U_B 28672  // 112 * 256

#define V_R 28672  // 112 * 256
#define V_G -24064 // -94 * 256
#define V_B -4608  // -18 * 256

#define CLAMP(x) ((x) < 0 ? 0 : ((x) > 255 ? 255 : x))

unsigned char *d_rgb24 = nullptr;
unsigned char *d_yuyv422 = nullptr;

__global__ void convert_rgb24_to_yuyv_cuda_kernel(const unsigned char *rgb24, unsigned char *yuyv422, unsigned int width, unsigned int height)
{
    int x = (blockIdx.x * blockDim.x + threadIdx.x) * 2;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height)
    {
        int index_rgb = (y * width + x) * 3;
        int index_yuyv = (y * width + x) * 2;

        unsigned char r0 = rgb24[index_rgb];
        unsigned char g0 = rgb24[index_rgb + 1];
        unsigned char b0 = rgb24[index_rgb + 2];
        unsigned char r1 = rgb24[index_rgb + 3];
        unsigned char g1 = rgb24[index_rgb + 4];
        unsigned char b1 = rgb24[index_rgb + 5];

        unsigned char y0 = CLAMP(((Y_R * r0 + Y_G * g0 + Y_B * b0) >> 16) + 16);
        unsigned char y1 = CLAMP(((Y_R * r1 + Y_G * g1 + Y_B * b1) >> 16) + 16);
        unsigned char u = CLAMP(((U_R * r0 + U_G * g0 + U_B * b0) >> 16) + 128);
        unsigned char v = CLAMP(((V_R * r0 + V_G * g0 + V_B * b0) >> 16) + 128);

        yuyv422[index_yuyv] = y0;
        yuyv422[index_yuyv + 1] = u;
        yuyv422[index_yuyv + 2] = y1;
        yuyv422[index_yuyv + 3] = v;
    }
}

void convert_rgb24_to_yuyv_cuda(const unsigned char *rgb24, unsigned char *yuyv422, unsigned int width, unsigned int height)
{
    static size_t size_rgb24 = width * height * 3 * sizeof(unsigned char);
    static size_t size_yuyv422 = width * height * 2 * sizeof(unsigned char);
    static dim3 blockSize(32, 16);
    static dim3 gridSize((width / 2 + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);
    if (d_rgb24 == nullptr)
    {
        hipMalloc((void **)&d_rgb24, size_rgb24);
        hipMalloc((void **)&d_yuyv422, size_yuyv422);
    }
    hipMemcpy(d_rgb24, rgb24, width * height * 3, hipMemcpyHostToDevice);
    convert_rgb24_to_yuyv_cuda_kernel<<<gridSize, blockSize>>>(d_rgb24, d_yuyv422, width, height);
    hipMemcpy(yuyv422, d_yuyv422, size_yuyv422, hipMemcpyDeviceToHost);
}

void cleanup_cuda_buffers()
{
    if (d_rgb24)
    {
        hipFree(d_rgb24);
        hipFree(d_yuyv422);
        d_rgb24 = nullptr;
        d_yuyv422 = nullptr;
    }
}

unsigned char *get_cuda_buffer(size_t size) 
{
    unsigned char *h_pinned = nullptr;
    hipHostMalloc((void **)&h_pinned, size);
    return h_pinned;
}

void free_cuda_buffer(unsigned char *h_pinned) 
{
    if (h_pinned) {
        hipHostFree(h_pinned);
    }
}
