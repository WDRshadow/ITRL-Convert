#include <hip/hip_runtime.h>

#include "hip_stream.h"
#include "bayerRG2rgb.h"
#include "rgb2yuyv.h"

bool is_bayer2yuyv_cuda_initialized = false;
int bayer2yuyv_stream_num_;
hipStream_t *bayer2yuyv_streams = nullptr;
unsigned int bayer2yuyv_width_;
unsigned int bayer2yuyv_height_;
unsigned int bayer2yuyv_block_height;
size_t bayer2yuyv_size_bayer_block;
size_t bayer2yuyv_size_rgb_block;
size_t bayer2yuyv_size_yuyv_block;
unsigned char *bayer2yuyv_d_bayer = nullptr;
unsigned char *bayer2yuyv_d_rgb = nullptr;
unsigned char *bayer2yuyv_d_yuyv = nullptr;

const dim3 bayer2yuyv_blockSize(32, 16);
dim3 bayer2yuyv_gridSize_1;
dim3 bayer2yuyv_gridSize_2;

void init_bayer2yuyv_cuda(unsigned int width, unsigned int height, int stream_num)
{
    if (is_bayer2yuyv_cuda_initialized)
    {
        cleanup_bayer2yuyv_cuda();
    }
    bayer2yuyv_width_ = width;
    bayer2yuyv_height_ = height;
    bayer2yuyv_stream_num_ = stream_num;
    bayer2yuyv_block_height = height / stream_num;
    bayer2yuyv_size_bayer_block = width * bayer2yuyv_block_height;
    bayer2yuyv_size_rgb_block = width * bayer2yuyv_block_height * 3;
    bayer2yuyv_size_yuyv_block = width * bayer2yuyv_block_height * 2;
    bayer2yuyv_gridSize_1 = dim3((width + bayer2yuyv_blockSize.x - 1) / bayer2yuyv_blockSize.x, (bayer2yuyv_block_height + bayer2yuyv_blockSize.y - 1) / bayer2yuyv_blockSize.y);
    bayer2yuyv_gridSize_2 = dim3((width / 2 + bayer2yuyv_blockSize.x - 1) / bayer2yuyv_blockSize.x, (bayer2yuyv_block_height + bayer2yuyv_blockSize.y - 1) / bayer2yuyv_blockSize.y);
    bayer2yuyv_streams = (hipStream_t *)malloc(stream_num * sizeof(hipStream_t));
    for (int i = 0; i < stream_num; i++)
    {
        hipStreamCreate(&bayer2yuyv_streams[i]);
    }
    hipMalloc((void **)&bayer2yuyv_d_bayer, width * height);
    hipMalloc((void **)&bayer2yuyv_d_rgb, width * height * 3);
    hipMalloc((void **)&bayer2yuyv_d_yuyv, width * height * 2);
    is_bayer2yuyv_cuda_initialized = true;
}

void bayer2yuyv_cuda(const unsigned char *bayer, unsigned char *yuyv)
{
    if (!is_bayer2yuyv_cuda_initialized)
    {
        return;
    }
    for (int i = 0; i < bayer2yuyv_stream_num_; i++)
    {
        hipMemcpyAsync(
            bayer2yuyv_d_bayer + i * bayer2yuyv_size_bayer_block,
            bayer + i * bayer2yuyv_size_bayer_block,
            bayer2yuyv_size_bayer_block,
            hipMemcpyHostToDevice,
            bayer2yuyv_streams[i]);

        bayerRG2rgb_kernel<<<bayer2yuyv_gridSize_1, bayer2yuyv_blockSize, 0, bayer2yuyv_streams[i]>>>(
            bayer2yuyv_d_bayer + i * bayer2yuyv_size_bayer_block,
            bayer2yuyv_d_rgb + i * bayer2yuyv_size_rgb_block,
            bayer2yuyv_width_,
            bayer2yuyv_block_height);

        rgb2yuyv_kernel<<<bayer2yuyv_gridSize_2, bayer2yuyv_blockSize, 0, bayer2yuyv_streams[i]>>>(
            bayer2yuyv_d_rgb + i * bayer2yuyv_size_rgb_block,
            bayer2yuyv_d_yuyv + i * bayer2yuyv_size_yuyv_block,
            bayer2yuyv_width_,
            bayer2yuyv_block_height);

        hipMemcpyAsync(
            yuyv + i * bayer2yuyv_size_yuyv_block,
            bayer2yuyv_d_yuyv + i * bayer2yuyv_size_yuyv_block,
            bayer2yuyv_size_yuyv_block,
            hipMemcpyDeviceToHost,
            bayer2yuyv_streams[i]);
    }
    hipDeviceSynchronize();
}

void cleanup_bayer2yuyv_cuda()
{
    if (!is_bayer2yuyv_cuda_initialized)
    {
        return;
    }
    for (int i = 0; i < bayer2yuyv_stream_num_; i++)
    {
        hipStreamDestroy(bayer2yuyv_streams[i]);
    }
    free(bayer2yuyv_streams);
    bayer2yuyv_streams = nullptr;
    hipFree(bayer2yuyv_d_bayer);
    bayer2yuyv_d_bayer = nullptr;
    hipFree(bayer2yuyv_d_rgb);
    bayer2yuyv_d_rgb = nullptr;
    hipFree(bayer2yuyv_d_yuyv);
    bayer2yuyv_d_yuyv = nullptr;
    is_bayer2yuyv_cuda_initialized = false;
}

unsigned char *get_cuda_buffer(size_t size)
{
    unsigned char *h_pinned = nullptr;
    hipHostMalloc((void **)&h_pinned, size);
    return h_pinned;
}

void free_cuda_buffer(unsigned char *h_pinned)
{
    if (h_pinned)
    {
        hipHostFree(h_pinned);
    }
}